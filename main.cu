#include "hip/hip_runtime.h"
#include "gdal.h"
#include "gdal_utils.h"
#include "cpl_conv.h"



// image resampling
__global__ void bicubicInterpolation(float * data, float * output, int width, int height, int outputw, int outputh){

}
// image resampling
__global__ void bilinearInterpolation(float * data, float * output, int width, int height, int outputw, int outputh){

}

// image resampling
__global__ void nearestNeighborInterpolation(float * data, float * output, int width, int height, int outputw, int outputh){

}

//convolution gaussian blur
__global__ void gaussianBlur(float * data, float * output, int width, int height){

}

//Convolution Sharpen
__global__ void sharpen(float * data, float * output, int width, int height){}

// Convolution https://en.wikipedia.org/wiki/Difference_of_Gaussians, Use shared memory here
__global__ void laplacianOfGaussian(float * data, float * output, int width, int height){
    
}

__global__ void ndvi(uint16_t* red, uint16_t* nir, float* output, int width, int height) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < width * height) {

        int denom = nir[tid] + red[tid];
        if (denom == 0 || nir[tid] == 0|| red[tid] == 0)
        {
            output[tid] = nanf("");
        }else{
            output[tid] = float (nir[tid] - red[tid])/ float (nir[tid] + red[tid]);

        }
        
        
    }
}

void cpuCalc(uint16_t* red, uint16_t* nir, float* output, int width, int height){

    for (int tid = 0; tid < width * height; tid++)
    {
        int denom = nir[tid] + red[tid];
        if (denom == 0 || nir[tid] == 0|| red[tid] == 0)
        {
            output[tid] = nanf("");
        }else{
            output[tid] = float (nir[tid] - red[tid])/ float (nir[tid] + red[tid]);

        }
    }
    
}



uint16_t* loadBand(const char* fileName, int width, int height) {
    GDALAllRegister();

    GDALDatasetH dataset = GDALOpen(fileName, GA_ReadOnly);
    GDALRasterBandH band = GDALGetRasterBand(dataset, 1);

    uint16_t* data = (uint16_t*)malloc(width * height * sizeof(uint16_t));
    GDALRasterIO(band, GF_Read, 0, 0, width, height, data, width, height, GDT_UInt16, 0, 0);

    GDALClose(dataset);
    return data;
}




void ndviImageCUDA(const char* rName, const char* nName,  const char* outputFileName)
{

    GDALAllRegister();
    GDALDatasetH dataSetInfo = GDALOpen(rName, GA_ReadOnly);
    int width = GDALGetRasterXSize(dataSetInfo);
    int height = GDALGetRasterYSize(dataSetInfo);



    uint16_t* redBand = loadBand(rName, width, height);
    uint16_t* nirBand = loadBand(nName, width, height);


    float* output = (float*)malloc(width * height * sizeof(float));


    uint16_t* d_redBand;

    uint16_t* d_nirBand;

    int bs = 256;
    int gs = (width * height + bs - 1) / bs;

    float * d_output;
    hipMalloc((void**)&d_redBand, width * height * sizeof(uint16_t));
    hipMalloc((void**)&d_nirBand, width * height * sizeof(uint16_t));

    hipMalloc((void**)&d_output, width * height * sizeof(float));


    hipMemcpy(d_redBand, redBand, width * height * sizeof(uint16_t), hipMemcpyHostToDevice);
    hipMemcpy(d_nirBand, nirBand, width * height * sizeof(uint16_t), hipMemcpyHostToDevice);

    

    ndvi<<<gs, bs>>>(d_redBand, d_nirBand, d_output, width, height);


    hipMemcpy(output, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    printf("NDVI CALC OK \n");

    free(redBand);
    free(nirBand);
    hipFree(d_redBand);
    hipFree(d_nirBand);
    hipFree(d_output);
    // Create a TIFF dataset to save the image

    GDALDriverH driver = GDALGetDriverByName("GTiff");
    GDALDatasetH tiffDataset = GDALCreate(driver, outputFileName, width, height, 1, GDT_Float32, NULL);

    
    double adfGeoTransform[6];
    GDALGetGeoTransform(dataSetInfo, adfGeoTransform);
    const char* spatialRef = GDALGetProjectionRef(dataSetInfo);
    GDALSetGeoTransform(tiffDataset, adfGeoTransform);
    GDALSetProjection(tiffDataset, spatialRef);
    GDALRasterBandH outBand = GDALGetRasterBand(tiffDataset, 1);
    GDALRasterIO(outBand, GF_Write, 0, 0, width, height, output, width, height, GDT_Float32, 0, 0);


    GDALClose(tiffDataset);
    GDALClose(dataSetInfo);


    free(output);


}




int main(int argc, char const *argv[])
{
    const char* redFile = "B02.tif";

    const char* nirFile = "B08.tif";


    ndviImageCUDA(redFile, nirFile, "ndvitest.tiff");

    return 0;
}
